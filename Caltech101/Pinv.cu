#include <hiprand.h>
#include <conio.h>
#include <iostream>
#include <hipblas.h>
#include "Pinv.h"



// using cublas matrix multiplication
 void ATA(hipblasHandle_t &handle,const double *A, double *C ,const int m, const int k) {
    

	
	int lda=m,ldb=m,ldc=k;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;
	 //create cuda stream
	
	hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, k, m, alpha, A, lda, A, ldb, beta, C, ldc);

 }
 
 
  void AATA(hipblasHandle_t &handle,const double *A,const double *B, double *C ,const int k, const int m) {
    

	
	int lda=k,ldb=m,ldc=k;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;
	 //create cuda stream
	
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, k, m, k, alpha, A, lda, B, ldb, beta, C, ldc);

 }

void gpu_inverse(hipblasHandle_t &handle, double *A,double *C,const int m){//Reference :https://stackoverflow.com/questions/37731103/cublas-matrix-inverse-much-slower-than-matlab
double** adL;
double** adC;
double* dL;
double* dC;
int* dLUPivots;
int* dLUInfo;

hipMalloc(&adL, sizeof(double*));
hipMalloc(&adC, sizeof(double*));
hipMalloc(&dL,  m * m * sizeof(double));
hipMalloc(&dC,  m * m * sizeof(double));
hipMalloc(&dLUPivots, m * sizeof(int));
hipMalloc(&dLUInfo, sizeof(int));
hipMemcpy(dL, A, m * m * sizeof(double), hipMemcpyDeviceToDevice);
hipMemcpy(adL, &dL, sizeof(double*), hipMemcpyHostToDevice);
hipMemcpy(adC, &dC, sizeof(double*), hipMemcpyHostToDevice);

hipblasDgetrfBatched(handle, m, adL, m, dLUPivots, dLUInfo, 1);
hipDeviceSynchronize();
hipblasDgetriBatched(handle, m, (const double **)adL, m, dLUPivots, adC, m, dLUInfo, 1);
hipDeviceSynchronize();
hipMemcpy(C, dC, m * m * sizeof(double), hipMemcpyDeviceToDevice);
hipFree(adC);
hipFree(adL);
hipFree(dC);
hipFree(dL);
hipFree(dLUInfo);
hipFree(dLUPivots);
}


void Pinv(double *d_A, double *d_B,const int m , const int k){


double *d_C;
int CSize = k*k;
hipMalloc((void**)&d_C, sizeof(double)*CSize) ;
hipblasHandle_t handle;
hipblasCreate(&handle);
	ATA(handle,d_A, d_C, m, k);
	
	gpu_inverse(handle,d_C,d_C,  k);
	
	AATA(handle,d_C, d_A,d_B, k, m);
	
	hipFree(d_A);
	hipFree(d_C);
	hipblasDestroy(handle);
	
	}
	





	
	
	
	
	
	
	