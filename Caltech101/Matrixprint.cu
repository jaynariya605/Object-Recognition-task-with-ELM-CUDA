#include <hiprand.h>
#include <conio.h>
#include <iostream>
#include <hipblas.h>
#include "MatAdd.h"
#include "Matrixprint.h"

void Matrixprint(double *a, int m, int n)
{
	for(int i=0;i<m;i++)
	{
		for(int j=0;j<n;j++)
		{
			printf("%f ",a[i+j*m]);
		}
		printf("\n");
	}
}