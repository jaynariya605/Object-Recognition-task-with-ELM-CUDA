#include <hiprand.h>
#include <conio.h>
#include <iostream>
#include <hipblas.h>
#include "MatAdd.h"

void MatAdd(const double *A, double *C ,const int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
	
	
	
	const double alf = 1;
	const double *alpha = &alf;
	
	hipblasDaxpy(handle, n,  alpha, A, 1,  C, 1);
	hipblasDestroy(handle);
	}
	
	

