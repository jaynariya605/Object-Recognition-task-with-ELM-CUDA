#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <conio.h>
#include <iostream>
#include <hipblas.h>
#include "RandomGPU.h"


// Reference : https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-hiprand-thrust/
// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void RandomGPU(double *A, int nr_rows_A, int nr_cols_A)
{
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
}