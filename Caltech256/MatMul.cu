#include <hiprand.h>
#include <conio.h>
#include <iostream>
#include <hipblas.h>
#include "MatMul.h"

void MatMul(const double *A, const double *B, double *C ,const int m, const int k, const int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
	
	
	int lda=m,ldb=k,ldc=m;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;
	
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	hipblasDestroy(handle);
	}
	
	
